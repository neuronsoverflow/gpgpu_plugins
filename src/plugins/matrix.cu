#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h> // checkCudaErrors()
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

typedef struct {
   int width;
   int height;
   float* elements;
} Matrix;

/******************************************************************************
* kernel for multiplying "C = A * B" on the Device
* kernel source: http://stackoverflow.com/a/18856054
******************************************************************************/
template <int BLOCK_SIZE>
__global__ void matrixMulCUDA(const Matrix A, const Matrix B, Matrix C)
{
   float cValue = 0;

   int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
   int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;

   __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
   __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

   for (int k = 0; k < (BLOCK_SIZE + A.width - 1) / BLOCK_SIZE; k++)
   {
      if (k * BLOCK_SIZE + threadIdx.x < A.width && row < A.height)
         As[threadIdx.y][threadIdx.x] = A.elements[row * A.width + k * BLOCK_SIZE + threadIdx.x];
      else
         As[threadIdx.y][threadIdx.x] = 0.0;

      if (k * BLOCK_SIZE + threadIdx.y < B.height && col < B.width)
         Bs[threadIdx.y][threadIdx.x] = B.elements[(k * BLOCK_SIZE + threadIdx.y) * B.width + col];
      else
         Bs[threadIdx.y][threadIdx.x] = 0.0;

      __syncthreads();

      for (int n = 0; n < BLOCK_SIZE; ++n)
         cValue += As[threadIdx.y][n] * Bs[n][threadIdx.x];

      __syncthreads();
   }

   if (row < C.height && col < C.width)
      C.elements[((blockIdx.y * blockDim.y + threadIdx.y) * C.width) + (blockIdx.x * blockDim.x) + threadIdx.x] = cValue;
}

/******************************************************************************
* displayTheMatrix()
* - displays the (host) matrix elements
******************************************************************************/
__host__ void displayTheMatrix(Matrix* matrix)
{
   if (matrix)
   {
      int row;
      int col;
      for (row = 0; row < matrix->height; ++row)
      {
         for (col = 0; col < matrix->width; ++col)
            printf("%3.0f ", matrix->elements[row * matrix->width + col]);
         printf("\n");
      }
   }
}

/******************************************************************************
* allocateDeviceMatrix
* - allocates the deviceMatrix with a copy of the host elements
******************************************************************************/
__host__ void allocateDeviceMatrix(const Matrix* hostMatrix, Matrix& deviceMatrix)
{
   deviceMatrix.width  = hostMatrix->width;
   deviceMatrix.height = hostMatrix->height;

   // allocate deviceMatrix.elements on the device
   int size = hostMatrix->width * hostMatrix->height * sizeof(float);
   if (hipMalloc(&deviceMatrix.elements, size) != hipSuccess)
   {
      printf("FAILED TO ALLOCATE MATRIX ELEMENTS ON DEVICE!\n");
      exit(EXIT_FAILURE);
   }

   // copy the matrix elements to the device
   if (hipMemcpy(deviceMatrix.elements, hostMatrix->elements, size, hipMemcpyHostToDevice) != hipSuccess)
   {
      printf("FAILED TO COPY MATRIX TO DEVICE!\n");
      exit(EXIT_FAILURE);
   }
}

/******************************************************************************
* freeDeviceMatrix()
* - deallocates the device matrix
******************************************************************************/
__host__ void freeDeviceMatrix(Matrix& matrix)
{
   if (matrix.elements)
   {
      hipError_t error = hipFree(matrix.elements);
      if (error != hipSuccess)
      {
         printf("hipFree returned error code %d, line(%d)\n", error, __LINE__);
         exit(EXIT_FAILURE);
      }
      matrix.elements = NULL;
   }
}

/******************************************************************************
* matrixMulOnDevice()
* - allocates the matrices on the device
* - call the kernel to compute C = A * B on the device
* - store the results back into C (host)
* - deallocate the device matrices
******************************************************************************/
__host__ void matrixMulOnDevice(const Matrix* A, const Matrix* B, Matrix* C)
{
   // query the Device and decide on the block size
   int devID = 0; // the default device ID
   hipError_t error;
   hipDeviceProp_t deviceProp;
   error = hipGetDevice(&devID);
   if (error != hipSuccess)
   {
      printf("hipGetDevice returned error code %d, line(%d)\n", error, __LINE__);
      exit(EXIT_FAILURE);
   }

   error = hipGetDeviceProperties(&deviceProp, devID);
   if (deviceProp.computeMode == hipComputeModeProhibited || error != hipSuccess)
   {
      printf("CUDA device ComputeMode is prohibited or failed to getDeviceProperties\n");
      return;
   }

   // Use a larger block size for Fermi and above (see compute capability)
   int block_size = (deviceProp.major < 2) ? 16 : 32;

   Matrix d_A;
   Matrix d_B;
   Matrix d_C;

   // allocate the matrices on the device
   allocateDeviceMatrix(A, d_A);
   allocateDeviceMatrix(B, d_B);
   allocateDeviceMatrix(C, d_C);

   // make sure that there are no errors...
   checkCudaErrors(hipPeekAtLastError());

   // setup the execution configuration
   /* dim3 dimGrid(1, 1); */
   /* dim3 dimBlock(A->width, A->width); */
   // see http://stackoverflow.com/a/19007136 for choice of Block and Grid size
   dim3 dimBlock(block_size, block_size);
   dim3 dimGrid;
   dimGrid.x = (C->width  + dimBlock.x - 1) / dimBlock.x;
   dimGrid.y = (C->height + dimBlock.y - 1) / dimBlock.y;

   //////// debug
   #ifdef DEBUG
   printf("dimBlock(%d, %d, %d)\n", dimBlock.x, dimBlock.y, dimBlock.z);
   printf("dimGrid(%d, %d, %d)\n", dimGrid.x, dimGrid.y, dimGrid.z);
   #endif

   // run the kernel
   if (block_size == 16)
      matrixMulCUDA<16><<< dimGrid, dimBlock >>>(d_A, d_B, d_C);
   else
      matrixMulCUDA<32><<< dimGrid, dimBlock >>>(d_A, d_B, d_C);

   // some kernel error checking...
   checkCudaErrors(hipPeekAtLastError());
   checkCudaErrors(hipDeviceSynchronize());

   // copy the matrix result from device to host
   int mem_size_C = C->height * C->width * sizeof(float);
   error = hipMemcpy(C->elements, d_C.elements, mem_size_C, hipMemcpyDeviceToHost);
   if (error != hipSuccess)
   {
      printf("hipMemcpy (C->elements, d_C.elements) returned error code %d, line(%d)\n", error, __LINE__);
      exit(EXIT_FAILURE);
   }

   // hipFree()
   freeDeviceMatrix(d_A);
   freeDeviceMatrix(d_B);
   freeDeviceMatrix(d_C);
}
