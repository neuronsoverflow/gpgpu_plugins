#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

typedef struct {
   int width;
   int height;
   float* elements;
} Matrix;

// Kernel that executes on the CUDA device
// http://stackoverflow.com/questions/18815489/cuda-tiled-matrix-matrix-multiplication-with-shared-memory-and-matrix-size-whic
template <int BLOCK_SIZE>
__global__ void matrixMulCUDA(Matrix* A, Matrix* B, Matrix* C)
{
   float CValue = 0;
   int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
   int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;

   __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
   __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

   for (int k = 0; k < (BLOCK_SIZE + A->width - 1)/BLOCK_SIZE; k++)
   {
      if (k*BLOCK_SIZE + threadIdx.x < A->width && row < A->height)
         As[threadIdx.y][threadIdx.x] = A->elements[row*A->width + k*BLOCK_SIZE + threadIdx.x];
      else
         As[threadIdx.y][threadIdx.x] = 0.0;

      if (k*BLOCK_SIZE + threadIdx.y < B->width && col < B->height)
         Bs[threadIdx.y][threadIdx.x] = B->elements[(k*BLOCK_SIZE + threadIdx.y)*B->height + col];
      else
         Bs[threadIdx.y][threadIdx.x] = 0.0;

      __syncthreads();

      for (int n = 0; n < BLOCK_SIZE; ++n)
         CValue += As[threadIdx.y][n] * Bs[n][threadIdx.x];

      __syncthreads();
   }

   if (row < C->height && col < C->width)
      C->elements[((blockIdx.y * blockDim.y + threadIdx.y)*C->width)+(blockIdx.x*blockDim.x)+threadIdx.x] = CValue;
}


void allocateDeviceMatrix(Matrix* hostMatrix, Matrix* deviceMatrix)
{
   if (hipMalloc(&deviceMatrix, sizeof(Matrix)) != hipSuccess)
      printf("FAILED TO ALLOCATE MATRIX ON DEVICE!\n");
   deviceMatrix->width  = hostMatrix->width;
   deviceMatrix->height = hostMatrix->height;

   int size = hostMatrix->width * hostMatrix->height * sizeof(float);
   if (hipMalloc(&(deviceMatrix->elements), size) != hipSuccess)
      printf("FAILED TO ALLOCATE MATRIX ELEMENTS ON DEVICE!\n");

   if (hipMemcpy(deviceMatrix, hostMatrix, size, hipMemcpyHostToDevice) != hipSuccess)
      printf("FAILED TO COPY MATRIX ELEMENTS TO DEVICE!\n");
}

// C = A * B
// this function allocates the matrices on the device, execute the kernel, and
// then store the result back in C
void matrixMulOnDevice(Matrix* A, Matrix* B, Matrix* C)
{
   // query the Device and decide on the block size
   int devID = 0; // the default device ID
   hipError_t error;
   hipDeviceProp_t deviceProp;
   error = hipGetDevice(&devID);
   if (error != hipSuccess)
   {
      printf("hipGetDevice returned error code %d, line(%d)\n", error, __LINE__);
   }

   error = hipGetDeviceProperties(&deviceProp, devID);

   if (deviceProp.computeMode == hipComputeModeProhibited ||
       error != hipSuccess)
   {
      // print error and return
   }

   // Use a larger block size for Fermi and above
   int block_size = (deviceProp.major < 2) ? 16 : 32;

   // TODO: veviry this dimsA, dimsB, threads, grid
   // not sure what 5*2 or 5*4 is coming from...
   dim3 dimsA(5*2*block_size, 5*2*block_size, 1);
   dim3 dimsB(5*4*block_size, 5*2*block_size, 1);
   dimsA.x = A->height;
   dimsA.y = A->width;
   dimsB.x = B->height;
   dimsB.y = B->width;

   Matrix* d_A = NULL;
   Matrix* d_B = NULL;
   Matrix* d_C = NULL;

   // allocate the matrices on the device
   allocateDeviceMatrix(A, d_A);
   allocateDeviceMatrix(B, d_B);
   allocateDeviceMatrix(C, d_C);

   // setup the execution configuration
   dim3 threads(block_size, block_size);
   dim3 grid(B->height / threads.x, A->width / threads.y);


   // compute the matrix multiplication by executing the kernel
   if (block_size == 16)
      matrixMulCUDA<16><<< grid, threads >>>(d_A, d_B, d_C);
   else
      matrixMulCUDA<32><<< grid, threads >>>(d_A, d_B, d_C);

   // copy the result from device to host
   int mem_size_C = C->height * C->width * sizeof(float);
   error = hipMemcpy(C->elements, d_C, mem_size_C, hipMemcpyDeviceToHost);
   if (error != hipSuccess)
   {
      printf("hipMemcpy (h_C,d_C) returned error code %d, line(%d)\n", error, __LINE__);
      exit(EXIT_FAILURE);
   }


}
