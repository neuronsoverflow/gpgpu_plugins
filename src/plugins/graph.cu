#include "hip/hip_runtime.h"
/******************************************************************************
*  The Graph Search algorithms follows the ones found in the paper:
*  "Accelerating Large Graph Algorithms on the GPU using CUDA" (HiPC 2007)
*  by Pawan Harish and P.J. Narayanan
*
*  This file implements:
*    - BFS
*    - SSSP
*    - APSP
******************************************************************************/
#include "graph.h"
// #include <float.h> // FLT_MAX
#include <hip/hip_runtime_api.h> // checkCudaErrors- NVIDIA_CUDA-6.0_Samples/common/inc
#include <limits.h>
#include <stdio.h>


////////////////////////// HELPER FUNCTIONS
// getNumEdges() - returns the number of edges for the vertex found in the
//                 given index
__host__ __device__ int getNumEdges(int index, int* vertices, int* edges,
                                    int numVertices, int numEdges)
{
   int count = vertices[(index + 1) % numVertices] - vertices[index];
   if (count < 0)
      count = numEdges - vertices[index];
   return count;
}

// getEdgeEndIndex() - given the starting index of the vertex, return the
//                     ending edge index for the given vertex
__host__ __device__ int getEdgeEndIndex(int index, int* vertices,
                               int numVertices, int numEdges)
{
   return (index + 1 < (numVertices)) ? vertices[index + 1] : numEdges;
}

/******************************************************************************
* checkDevice()
******************************************************************************/
__host__ int checkDevice()
{
   // query the Device and decide on the block size
   int devID = 0; // the default device ID
   hipError_t error;
   hipDeviceProp_t deviceProp;
   error = hipGetDevice(&devID);
   if (error != hipSuccess)
   {
      printf("CUDA Device not ready or not supported\n");
      printf("%s: hipGetDevice returned error code %d, line(%d)\n", __FILE__, error, __LINE__);
      exit(EXIT_FAILURE);
   }

   error = hipGetDeviceProperties(&deviceProp, devID);
   if (deviceProp.computeMode == hipComputeModeProhibited || error != hipSuccess)
   {
      printf("CUDA device ComputeMode is prohibited or failed to getDeviceProperties\n");
      return EXIT_FAILURE;
   }

   // Use a larger block size for Fermi and above (see compute capability)
   return (deviceProp.major < 2) ? 16 : 32;
}

////////////////////////// KERNEL FUNCTIONS

/******************************************************************************
*  CUDA_BFS_KERNEL (Va, Ea, Fa, Xa, Ca)
*
*  This algorithm follows the one found in the paper
*    "Accelerating Large Graph Algorithms on the GPU using CUDA"
*    by Pawan Harish and P.J. Narayanan
*
*  Algortithm:
*  1: tid ← getThreadID
*  2: if Fa [tid] then
*  3:   Fa [tid] ← false, Xa [tid] ← true
*  4:   for all neighbors nid of tid do
*  5:     if NOT Xa [nid] then
*  6:       Ca [nid] ← Ca [tid]+1
*  7:       Fa [nid] ← true
*  8:     end if
*  9:   end for
*  10:end if
******************************************************************************/
__global__ void bfsCUDA(int* vertices, int* edges, bool* frontier,
                        bool* visited, int* cost, bool* keepRunning,
                        int numVertices, int numEdges)
{
   int tid = blockIdx.x * blockDim.x + threadIdx.x; // 1
   if (tid < numVertices && frontier[tid]) // 2
   {
      // 3
      frontier[tid] = false;
      visited[tid] = true;
      int edgeStart = vertices[tid];
      int edgeEnd = getEdgeEndIndex(tid, vertices, numVertices, numEdges);

      // 4
      // "If true, it fetches its cost from the cost array C and updates all
      // the costs of its neighbors if more than its own cost plus one using
      // the edge list E."

      // "The vertex removes its own entry from the frontier array F and adds
      // to the visited array Xa . It also adds its neighbors to the frontier
      // array if the neighbor is not already visited."
      for(int edge = edgeStart; edge < edgeEnd; edge++)
      {
         int nid = edges[edge];
         if (!visited[nid])
         {
            cost[nid] = cost[tid] + 1;
            frontier[nid] = true;
            *keepRunning = true;
         }
      }
   }
}

/******************************************************************************
* CUDA_SSSP_KERNEL1 (Va, Ea, Wa, Ma, Ca, Ua)
*
* 1: tid ← getThreadID
* 2: if Ma [tid] then
* 3:   Ma [tid] ← false
* 4:   for all neighbors nid of tid do
* 5:     if Ua [nid]> Ca [tid]+Wa [edge] then
* 6:       Ua [nid] ← Ca [tid]+Wa [edge]
* 7:     end if
* 8:   end for
* 9: end if
******************************************************************************/
__global__ void ssspCUDA1(int* vertices, int* edges, int* weights,
                          int* mask, int* cost, int* updatingCost,
                          // bool* mask, float* cost, float* updatingCost,
                          int numVertices, int numEdges)
{
   unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
   if (tid < numVertices &&  mask[tid])
   {
      mask[tid] = 0;

      int edgeStart = vertices[tid];
      int edgeEnd = getEdgeEndIndex(tid, vertices, numVertices, numEdges);

      // edge = index of the neighbor in the edges array and in the weights arr
      for(int edge = edgeStart; edge < edgeEnd; edge++)
      {
         int nid = edges[edge]; // neighbor vertex index
         if (updatingCost[nid] > (cost[tid] + weights[edge]))
         {
            // updatingCost[nid] = (cost[tid] + weights[edge]);
            atomicMin(&updatingCost[nid], cost[tid] + weights[edge]);
         }
      }
   }
}

/******************************************************************************
* CUDA_SSSP_KERNEL2 (Va, Ea, Wa, Ma, Ca, Ua)
*
* 1: tid ← getThreadID
* 2: if Ca [tid] > Ua [tid] then
* 3:   Ca [tid] ← Ua [tid]
* 4:   Ma [tid] ← true
* 5: end if
* 6: Ua [tid] ← Ca [tid]
******************************************************************************/
__global__ void ssspCUDA2(int* vertices, int* edges, int* weights,
                          int* mask, int* cost, int* updatingCost,
                          int numVertices, int numEdges, bool* keepRunning)
{
   int tid = blockIdx.x * blockDim.x + threadIdx.x;
   if (tid < numVertices)
   {
      if (cost[tid] > updatingCost[tid])
      {
         // cost[tid] = updatingCost[tid];
         atomicMin(&cost[tid], updatingCost[tid]);
         // atomicAdd(&mask[tid], 1); // mask[tid] = true;
         atomicMax(&mask[tid], 1); // mask[tid] = true;
         *keepRunning = true;
      }
      updatingCost[tid] = cost[tid];
   }
}

/******************************************************************************
*  CUDA_BFS (Graph G(V, E), Source Vertex S)
*
*  1. Create vertex array Va from all vertices and edge Array Ea from all
*     edges in G(V, E),
*  2. Create frontier array Fa , visited array X a and cost array Ca of size V.
*  3. Initialize Fa , Xa to false and Ca to ∞
*  4. Fa [S] ← true, Ca [S] ← 0
*  5. while Fa not Empty do
*  6.   for each vertex V in parallel do
*  7.     Invoke CUDA BFS KERNEL(Va, Ea, Fa, Xa, Ca) on the grid.
*  8.   end for
*  9. end while
*
*  "The BFS problem is, given an undirected, unweighted graph G(V, E) and a
*     source vertex S, find the minimum number of edges needed to reach every
*     vertex V in G from source vertex S."
*
* PARAMETERS:
* graph - The Graph Data - Graph G(V,E)
* sourceIndex - the index for the vertex/node to perform the BFS on.
* h_Cost - an array of the size of the number of vertices.
*          The final BFS result will be saved here, with the BFS costs to each
*          node.
******************************************************************************/
__host__ void bfsOnDevice(GraphData* graph, int sourceIndex, int* h_Cost)
{
   if (!graph || sourceIndex < 0 || sourceIndex >= graph->vertexCount ||
       !h_Cost)
   {
      printf("Failed to run BFS on the Device\n");
      return;
   }

   // HOST graph data
   bool* h_Frontier; // Fa (frontier array)
   bool* h_Visited;  // Xa (visited array)
                     // h_Cost = Ca (cost array)
   int numVertices = graph->vertexCount;
   int sizeVBool = sizeof(bool) * numVertices;
   int sizeVInt  = sizeof(int) * numVertices;

   // 2, 3
   h_Frontier = (bool*) calloc(numVertices, sizeof(bool));
   h_Visited  = (bool*) calloc(numVertices, sizeof(bool));
   h_Frontier[sourceIndex] = true;
   memset(h_Cost, -1, sizeVInt);

   // 4
   h_Visited[sourceIndex] = 1;
   h_Cost[sourceIndex] = 0;

   // DEVICE graph data
   int blockSize = checkDevice();
   int*  d_Vertices; // vertices / nodes
   int*  d_Edges;    // edges
   bool* d_Frontier; // frontier / mask
   bool* d_Visited;  // visited
   int*  d_Cost;     // cost
   bool* d_keepRunning;

   // 1 - copy the vertices and edges: HOST -> DEVICE (GPU)
   checkCudaErrors(hipMalloc(&d_Vertices, sizeVInt));
   checkCudaErrors(hipMalloc(&d_Edges, sizeof(int) * graph->edgeCount));
   checkCudaErrors(hipMemcpy(d_Vertices, graph->vertexArray, sizeVInt,
                              hipMemcpyHostToDevice));
   checkCudaErrors(hipMemcpy(d_Edges, graph->edgeArray,
                              sizeof(int) * graph->edgeCount,
                              hipMemcpyHostToDevice));

   // 2 - copy the frontier, visited and cost arrays: HOST -> DEVICE
   checkCudaErrors(hipMalloc(&d_Frontier, sizeVBool));
   checkCudaErrors(hipMalloc(&d_Visited, sizeVBool));
   checkCudaErrors(hipMalloc(&d_Cost, sizeof(int)  * numVertices));
   checkCudaErrors(hipMalloc(&d_keepRunning, sizeof(bool)));
   checkCudaErrors(hipMemcpy(d_Frontier, h_Frontier, sizeVBool,
                              hipMemcpyHostToDevice));
   checkCudaErrors(hipMemcpy(d_Visited, h_Visited, sizeVBool,
                              hipMemcpyHostToDevice));
   checkCudaErrors(hipMemcpy(d_Cost, h_Cost, sizeVInt,
                              hipMemcpyHostToDevice));

   // set up the execution parameters
   // 1 thread for each vertex
   dim3 dimBlock(blockSize);
   dim3 dimGrid(ceil(numVertices / (double) dimBlock.x));

   #ifdef DEBUG
   printf("dimBlock(%d, %d, %d)\n", dimBlock.x, dimBlock.y, dimBlock.z);
   printf("dimGrid(%d, %d, %d)\n", dimGrid.x, dimGrid.y, dimGrid.z);
   #endif

   bool keepRunning = true;

   // 4.loop while the Frontier array is not empty
   while (keepRunning)
   {
      keepRunning = false;
      // update the "keepRunning" boolean variable: HOST --> DEVICE
      checkCudaErrors(hipMemcpy(d_keepRunning, &keepRunning, sizeof(bool),
                                 hipMemcpyHostToDevice));
      // call the KERNEL
      bfsCUDA<<< dimGrid, dimBlock >>>(d_Vertices, d_Edges, d_Frontier,
                                       d_Visited, d_Cost, d_keepRunning,
                                       numVertices, graph->edgeCount);
      checkCudaErrors(hipPeekAtLastError());   // check for Kernel errors
      checkCudaErrors(hipDeviceSynchronize()); // block the CPU until GPU done

      // update the "keepRunning" boolean variable: HOST <-- DEVICE
      checkCudaErrors(hipMemcpy(&keepRunning, d_keepRunning, sizeof(bool),
                                 hipMemcpyDeviceToHost));
   }

   // Final BFS costs result will be in d_cost -> h_cost
   checkCudaErrors(hipMemcpy(h_Cost, d_Cost, sizeVInt,
                              hipMemcpyDeviceToHost));

   // release the krakens!
   checkCudaErrors(hipFree(d_Vertices));
   checkCudaErrors(hipFree(d_Edges));
   checkCudaErrors(hipFree(d_Frontier));
   checkCudaErrors(hipFree(d_Visited));
   checkCudaErrors(hipFree(d_Cost));
   checkCudaErrors(hipFree(d_keepRunning));

   free(h_Frontier);
   free(h_Visited);
}

/******************************************************************************
*  CUDA_SSSP (Graph G(V, E,W ), Source Vertex S)
*
* 1: Create vertex array Va , edge array Ea and weight array Wa from G(V, E, W)
* 2: Create mask array Ma , cost array Ca and Updating cost array Ua of size V
* 3: Initialize mask Ma to false, cost array Ca and Updating cost array Ua to ∞
* 4: Ma [S] ← true, Ca [S] ← 0, Ua [S] ← 0
* 5: while Ma not Empty do
* 6:   for each vertex V in parallel do
* 7:     Invoke CUDA SSSP KERNEL1(Va, Ea, Wa, Ma, Ca, Ua) on the grid
* 8:     Invoke CUDA SSSP KERNEL2(Va, Ea, Wa, Ma, Ca, Ua) on the grid
* 9:   end for
*10: end while
*
*  "Single source shortest path (SSSP) problem is, given weighted graph
*     G(V,E,W) with positive weights and a source vertex S, find the smallest
*     combined weight of edges that is required to reach every vertex V from
*     source vertex S"
******************************************************************************/
__host__ void ssspOnDevice(GraphData* graph, int sourceIndex, int* h_Cost)
{
   if (!graph || !h_Cost ||
       sourceIndex < 0   || sourceIndex >= graph->vertexCount)
   {
      printf("Failed to run SSSP on the Device\n");
      return;
   }

   // HOST graph data
   int* h_Mask;         // Ma (mask array)
   int* h_UpdatingCost; // Ua (updating cost array)

   int numVertices = graph->vertexCount;
   // NOTE: the cost array and updating cost array should have been float*
   //       Also, the mask array should have been bool* or char*
   //       However, atomicMin() and atomicMax() doesn't play well w/ these
   //         data-types, so we're using integers...
   // int sizeVFloat  = sizeof(float) * numVertices;
   // int sizeVBool   = sizeof(bool)  * numVertices;
   int sizeVInt    = sizeof(int)   * numVertices;

   h_Mask = (int*) malloc (sizeVInt);
   h_UpdatingCost = (int*) malloc (sizeVInt);

   // 3.
   memset(h_Mask, 0, sizeVInt);
   for (int i = 0; i < numVertices; i++)
   {
      h_Cost[i] = INT_MAX;  // use FLT_MAX if float*
      h_UpdatingCost[i] = INT_MAX;
   }

   // 4.
   h_Mask[sourceIndex] = 1;
   h_Cost[sourceIndex] = 0;
   h_UpdatingCost[sourceIndex] = 0;

   // DEVICE graph data
   int    blockSize = checkDevice();
   int*   d_Vertices;
   int*   d_Edges;
   int*   d_Weights;
   int*   d_Mask;
   int*   d_Cost;
   int*   d_UpdatingCost;
   bool*  d_keepRunning;

   // 1 - copy the vertices, edges and weights: HOST --> DEVICE (GPU)
   checkCudaErrors(hipMalloc(&d_Vertices, sizeVInt));
   checkCudaErrors(hipMalloc(&d_Edges,   sizeof(int) * graph->edgeCount));
   checkCudaErrors(hipMalloc(&d_Weights, sizeof(int) * graph->edgeCount));
   checkCudaErrors(hipMemcpy(d_Vertices, graph->vertexArray, sizeVInt,
                              hipMemcpyHostToDevice));
   checkCudaErrors(hipMemcpy(d_Edges, graph->edgeArray,
                              sizeof(int) * graph->edgeCount,
                              hipMemcpyHostToDevice));
   checkCudaErrors(hipMemcpy(d_Weights, graph->weightArray,
                              sizeof(int) * graph->edgeCount,
                              hipMemcpyHostToDevice));

   // 2 - copy the mask, cost, updating cost arrays: HOST --> DEVICE
   checkCudaErrors(hipMalloc(&d_Mask, sizeVInt));
   checkCudaErrors(hipMalloc(&d_Cost, sizeVInt));
   checkCudaErrors(hipMalloc(&d_UpdatingCost, sizeVInt));
   checkCudaErrors(hipMalloc(&d_keepRunning, sizeof(bool)));
   checkCudaErrors(hipMemcpy(d_Mask, h_Mask, sizeVInt,
                              hipMemcpyHostToDevice));
   checkCudaErrors(hipMemcpy(d_Cost, h_Cost, sizeVInt,
                              hipMemcpyHostToDevice));
   checkCudaErrors(hipMemcpy(d_UpdatingCost, h_UpdatingCost, sizeVInt,
                              hipMemcpyHostToDevice));

   // set up the execution parameters
   // 1 thread for each vertex
   dim3 dimBlock(blockSize);
   dim3 dimGrid(ceil(numVertices / (double) dimBlock.x));

   //////// debug
   #ifdef DEBUG
   printf("numVertices: %d\n", numVertices);
   printf("dimBlock(%d, %d, %d)\n", dimBlock.x, dimBlock.y, dimBlock.z);
   printf("dimGrid(%d, %d, %d)\n", dimGrid.x, dimGrid.y, dimGrid.z);
   #endif

   bool keepRunning = true;

   // 5. loop while the Mask array is not empty
   while (keepRunning)
   {
      keepRunning = false;
      checkCudaErrors(hipMemcpy(d_keepRunning, &keepRunning, sizeof(bool),
                                 hipMemcpyHostToDevice));
      // call the KERNELs
      ssspCUDA1<<< dimGrid, dimBlock >>>(d_Vertices, d_Edges, d_Weights,
                                         d_Mask, d_Cost, d_UpdatingCost,
                                         numVertices, graph->edgeCount);
      ssspCUDA2<<< dimGrid, dimBlock >>>(d_Vertices, d_Edges, d_Weights,
                                         d_Mask, d_Cost, d_UpdatingCost,
                                         numVertices, graph->edgeCount,
                                         d_keepRunning);
      // //////////////// DEBUG
      #ifdef DEBUG

      // copy back the current cost, ucost, mask
      checkCudaErrors(hipMemcpy(h_Cost, d_Cost, sizeVInt,
                                 hipMemcpyDeviceToHost));
      checkCudaErrors(hipMemcpy(h_UpdatingCost, d_UpdatingCost, sizeVInt,
                                 hipMemcpyDeviceToHost));
      checkCudaErrors(hipMemcpy(h_Mask, d_Mask, sizeVInt,
                                 hipMemcpyDeviceToHost));

      // display the current cost, ucost, umask
      printf("after a iteration: \n");
      printf("cost:  ");
      for (int i = 0; i < numVertices; i++)
      {
         printf("%5d ", h_Cost[i]);
      }
      printf("\n");
      printf("ucost: ");
      for (int i = 0; i < numVertices; i++)
      {
         printf("%5d ", h_UpdatingCost[i]);
      }
      printf("\n");
      printf("mask:  ");
      for (int i = 0; i < numVertices; i++)
      {
         printf("%5d ", h_Mask[i]);
      }
      printf("\n");
      #endif
      //////////////// END DEBUG

      // update the "keepRunning" boolean variable: HOST <- DEVICE
      checkCudaErrors(hipMemcpy(&keepRunning, d_keepRunning, sizeof(bool),
                                 hipMemcpyDeviceToHost));
   }

   // copy the computed costs back to the host
   checkCudaErrors(hipMemcpy(h_Cost, d_Cost, sizeVInt,
                              hipMemcpyDeviceToHost));
   // free the memory
   checkCudaErrors(hipFree(d_Vertices));
   checkCudaErrors(hipFree(d_Edges));
   checkCudaErrors(hipFree(d_Weights));
   checkCudaErrors(hipFree(d_Mask));
   checkCudaErrors(hipFree(d_Cost));
   checkCudaErrors(hipFree(d_UpdatingCost));
   checkCudaErrors(hipFree(d_keepRunning));

   free(h_Mask);
   free(h_UpdatingCost);
}

/******************************************************************************
*  ASPS_USING_SSSP(G(V,E,W))
*
* 1:  Create vertex array Va , edge array Ea , weight array Wa from G(V,E,W),
* 2:  Create mask array Ma , cost array Ca and updating cost array Ua of size V
* 3:  for S from 1 to V do
* 4:    Ma [S] ← true
* 6:    Ca [S] ← 0
* 7:    while Ma not Empty do
* 8:      for each vertex V in parallel do
* 9:        Invoke CUDA SSSP KERNEL1(Va, Ea, Wa, Ma, Ca, Ua) on the grid
* 10:       Invoke CUDA SSSP KERNEL2(Va, Ea, Wa, Ma, Ca, Ua) on the grid
* 11:     end for
* 12:   end while
* 13: end for
******************************************************************************/
__host__ void apspOnDevice(GraphData* graph, int** h_Costs)
{
   if (!graph || !h_Costs)
   {
      printf("Failed to run APSP on the Device\n");
      return;
   }

   // HOST graph data
   int* h_Infinity;

   int numVertices = graph->vertexCount;
   int sizeVInt    = sizeof(int) * numVertices;

   h_Infinity = (int*) malloc (sizeVInt);
   for (int i = 0; i < numVertices; i++)
   {
      h_Infinity[i] = INT_MAX;
   }

   // DEVICE graph data
   int   blockSize = checkDevice();
   int*  d_Vertices;
   int*  d_Edges;
   int*  d_Weights;
   int*  d_Mask;
   int*  d_Cost;
   int*  d_UpdatingCost;
   bool* d_keepRunning;

   // 1 - copy the vertices, edges and weights: HOST --> DEVICE (GPU)
   checkCudaErrors(hipMalloc(&d_Vertices, sizeVInt));
   checkCudaErrors(hipMalloc(&d_Edges,   sizeof(int)   * graph->edgeCount));
   checkCudaErrors(hipMalloc(&d_Weights, sizeof(int) * graph->edgeCount));
   checkCudaErrors(hipMemcpy(d_Vertices, graph->vertexArray, sizeVInt,
                              hipMemcpyHostToDevice));
   checkCudaErrors(hipMemcpy(d_Edges, graph->edgeArray,
                              sizeof(int) * graph->edgeCount,
                              hipMemcpyHostToDevice));
   checkCudaErrors(hipMemcpy(d_Weights, graph->weightArray,
                              sizeof(int) * graph->edgeCount,
                              hipMemcpyHostToDevice));

   // 2 - allocate the mask, cost, updating cost arrays on the DEVICE
   checkCudaErrors(hipMalloc(&d_Mask, sizeVInt));
   checkCudaErrors(hipMalloc(&d_Cost, sizeVInt));
   checkCudaErrors(hipMalloc(&d_UpdatingCost, sizeVInt));
   checkCudaErrors(hipMalloc(&d_keepRunning, sizeof(bool)));

   // set up the execution parameters
   // 1 thread for each vertex
   dim3 dimBlock(blockSize);
   dim3 dimGrid(ceil(numVertices / (double) dimBlock.x));

   for (int vertex = 0; vertex < numVertices; vertex++)
   {
      // reset the mask, cost, updating cost arrays on DEVICE
      checkCudaErrors(hipMemset(d_Mask, 0, sizeVInt));
      checkCudaErrors(hipMemcpy(d_Cost, h_Infinity, sizeVInt,
                                 hipMemcpyHostToDevice));
      checkCudaErrors(hipMemcpy(d_UpdatingCost, h_Infinity, sizeVInt,
                                 hipMemcpyHostToDevice));

      // 4, 5
      // initMaskAndCost<<< 1, 1 >>>(d_Mask, d_Cost, vertex);
      checkCudaErrors(hipMemset(d_Mask + vertex, true, sizeof(int)));
      checkCudaErrors(hipMemset(d_Cost + vertex, 0, sizeof(int)));

      bool keepRunning = true;
      while (keepRunning)
      {
         keepRunning = false;
         checkCudaErrors(hipMemcpy(d_keepRunning, &keepRunning, sizeof(bool),
                                    hipMemcpyHostToDevice));
         // call the KERNELs
         ssspCUDA1<<< dimGrid, dimBlock >>>(d_Vertices, d_Edges, d_Weights,
                                            d_Mask, d_Cost, d_UpdatingCost,
                                            numVertices, graph->edgeCount);
  // checkCudaErrors(hipPeekAtLastError());   // check for Kernel errors
  // checkCudaErrors(hipDeviceSynchronize()); // block the CPU until GPU done
         ssspCUDA2<<< dimGrid, dimBlock >>>(d_Vertices, d_Edges, d_Weights,
                                            d_Mask, d_Cost, d_UpdatingCost,
                                            numVertices, graph->edgeCount,
                                            d_keepRunning);
         checkCudaErrors(hipPeekAtLastError());   // check for Kernel errors
         checkCudaErrors(hipDeviceSynchronize()); // block the CPU until GPU done

         // update the "keepRunning" boolean variable: HOST <- DEVICE
         checkCudaErrors(hipMemcpy(&keepRunning, d_keepRunning, sizeof(bool),
                                    hipMemcpyDeviceToHost));
      }

      checkCudaErrors(hipMemcpy(h_Costs[vertex], d_Cost, sizeVInt,
                                 hipMemcpyDeviceToHost));
   }

   // free the memory
   checkCudaErrors(hipFree(d_Vertices));
   checkCudaErrors(hipFree(d_Edges));
   checkCudaErrors(hipFree(d_Weights));
   checkCudaErrors(hipFree(d_Mask));
   checkCudaErrors(hipFree(d_Cost));
   checkCudaErrors(hipFree(d_UpdatingCost));
   checkCudaErrors(hipFree(d_keepRunning));

   free(h_Infinity);
}
