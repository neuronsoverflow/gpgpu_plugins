#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h> // checkCudaErrors- NVIDIA_CUDA-6.0_Samples/common/inc

typedef unsigned long long int uint64_t;

/******************************************************************************
* kernel that initializes the 1st couple of values in the primes array.
******************************************************************************/
__global__ static void sieveInitCUDA(char* primes)
{
   primes[0] = 1; // value of 1 means the number is NOT prime
   primes[1] = 1; // numbers "0" and "1" are not prime numbers
}

/******************************************************************************
* kernel for sieving the even numbers starting at 4.
******************************************************************************/
__global__ static void sieveEvenNumbersCUDA(char* primes, uint64_t max)
{
   uint64_t index = blockIdx.x * blockDim.x *2 + threadIdx.x + threadIdx.x + 4;
   if (index < max)
      primes[index] = 1; // mark off the even numbers
}

/******************************************************************************
* kernel for finding prime numbers using the sieve of eratosthenes
* - primes: an array of bools. initially all numbers are set to "0".
*           A "0" value means that the number at that index is prime.
* - max: the max size of the primes array
* - maxRoot: the sqrt of max (the other input). we don't wanna make all threads
*   compute this over and over again, so it's being passed in
******************************************************************************/
__global__ static void sieveOfEratosthenesCUDA(char *primes, uint64_t max,
                                               const uint64_t maxRoot)
{
   // get the starting index, sieve only odds starting at 3
   // block 0: 3,   5,  7,  9, 11, 13, ...,  65
   // block 1: 67, 69, 71, 73, 75, 77, ..., 129
   uint64_t index = blockIdx.x * blockDim.x *2 + threadIdx.x + threadIdx.x + 3;

   // make sure index won't go out of bounds, also don't start the execution
   // on numbers that are already composite
   if (index <= maxRoot && primes[index] == 0)
   {
      // mark off the composite numbers
      for (int j = index * index; j < max; j += index)
      {
         primes[j] = 1;
      }
   }
}

/******************************************************************************
* checkDevice()
******************************************************************************/
__host__ int checkDevice()
{
   // query the Device and decide on the block size
   int devID = 0; // the default device ID
   hipError_t error;
   hipDeviceProp_t deviceProp;
   error = hipGetDevice(&devID);
   if (error != hipSuccess)
   {
      printf("CUDA Device not ready or not supported\n");
      printf("%s: hipGetDevice returned error code %d, line(%d)\n", __FILE__, error, __LINE__);
      exit(EXIT_FAILURE);
   }

   error = hipGetDeviceProperties(&deviceProp, devID);
   if (deviceProp.computeMode == hipComputeModeProhibited || error != hipSuccess)
   {
      printf("CUDA device ComputeMode is prohibited or failed to getDeviceProperties\n");
      return EXIT_FAILURE;
   }

   // Use a larger block size for Fermi and above (see compute capability)
   return (deviceProp.major < 2) ? 16 : 32;
}

/******************************************************************************
* genPrimesOnDevice
* - inputs: limit - the largest prime that should be computed
*           primes - an array of size [limit], initialized to 0
******************************************************************************/
__host__ void genPrimesOnDevice(char* primes, uint64_t max)
{
   int blockSize = checkDevice();
   if (blockSize == EXIT_FAILURE)
      return;

   char* d_Primes = NULL;
   int sizePrimes = sizeof(char) * max;
   uint64_t maxRoot = sqrt(max);

   // allocate the primes on the device and set them to 0
   checkCudaErrors(hipMalloc(&d_Primes, sizePrimes));
   checkCudaErrors(hipMemset(d_Primes, 0, sizePrimes));

   // make sure that there are no errors...
   checkCudaErrors(hipPeekAtLastError());

   // setup the execution configuration
   dim3 dimBlock(blockSize);
   // dim3 dimGrid((maxRoot + dimBlock.x) / dimBlock.x);
   dim3 dimGrid(ceil(
                   (maxRoot + dimBlock.x) / (double) dimBlock.x) / (double) 2);
   dim3 dimGridEvens(ceil(
                        (max + dimBlock.x) / (double) dimBlock.x) / (double) 2);

   // if for some reason, the user wants to find primes below 32 @.@ ...
   if (dimGridEvens.x < 1)
      dimGridEvens.x = 1; // make sure this won't be a '0'

   //////// debug
   #ifdef DEBUG
   printf("max: %llu\n", max);
   printf("maxRoot: %llu\n", maxRoot);
   printf("dimBlock(%d, %d, %d)\n", dimBlock.x, dimBlock.y, dimBlock.z);
   printf("dimGrid(%d, %d, %d)\n", dimGrid.x, dimGrid.y, dimGrid.z);
   printf("dimGridEvens(%d, %d, %d)\n", dimGridEvens.x, dimGridEvens.y, dimGridEvens.z);
   #endif

   // call the kernel
   // NOTE: no need to synchronize after each kernel
   // http://stackoverflow.com/a/11889641/2261947

   // NOTE: not sure why the 1st memset won't work. That would replace the
   //       sieveInitCUDA kernel...
   // checkCudaErrors(hipMemset(primes, 0, sizeof(char)));
   // checkCudaErrors(hipMemset(primes + 1, 1, sizeof(char)));
   sieveInitCUDA<<<1, 1>>>(d_Primes); // launch a single thread to initialize
   sieveEvenNumbersCUDA<<<dimGridEvens, dimBlock>>>(d_Primes, max);
   sieveOfEratosthenesCUDA<<<dimGrid, dimBlock>>>(d_Primes, max, maxRoot);

   // check for kernel errors
   checkCudaErrors(hipPeekAtLastError());
   checkCudaErrors(hipDeviceSynchronize());

   // copy the results back
   checkCudaErrors(hipMemcpy(primes, d_Primes, sizePrimes, hipMemcpyDeviceToHost));

   // no memory leaks
   checkCudaErrors(hipFree(d_Primes));
}
